#include "hip/hip_runtime.h"
#include "lpRgpu.cuh"
#include "main_kernels.cuh"
#include "simple_kernels.cuh"
#include "callerr.cuh"
lpRgpu::lpRgpu(int* N,int m0,float* fZ,int m1,int* st,int m2,int* stadj,int m3,int* idthetatx,int m4,int* idrhotx,int m5,int* idthetatauq,int m6,int* idrhotauq,int m7,float* dthetatx,int m8,
		float* drhotx,int m9,float* dthetatauq,int m10,float* drhotauq,int m11,float* emul,int m12,float* cosmul,int m13,float* J,int m14,int* reorids,int m15,int* reoridsadj,int m16)
{
	Nt=N[0];Nx=N[1];Nq=N[2];Ntau=N[3];Ntheta=N[4];Nrho=N[5];ni=N[6];
	Ntheta_R2C=(int)(Ntheta/2.0)+1;
	add=2;
	err=hipMalloc((void **)&dftx, Nt*Nx*ni*sizeof(float));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&dftauq, Ntau*Nq*ni*sizeof(float));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&dfl, (Ntheta+2*add)*(Nrho+2*add)*ni*sizeof(float));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&dflc, Ntheta_R2C*Nrho*ni*sizeof(float2));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&dfZ, Ntheta_R2C*Nrho*sizeof(float2));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&dst, (Ntheta+2*add)*(Nrho+2*add)*ni*sizeof(int));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&dstadj, (Ntheta+2*add)*(Nrho+2*add)*ni*sizeof(int));if (err!=0) callErr(hipGetErrorString(err));
	
	err=hipMalloc((void **)&ddthetatx, Nt*Nx*sizeof(float));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&ddrhotx, Nt*Nx*sizeof(float));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&ddthetatauq, Ntau*Nq*sizeof(float));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&ddrhotauq, Ntau*Nq*sizeof(float));if (err!=0) callErr(hipGetErrorString(err));

	err=hipMalloc((void **)&didthetatx, Nt*Nx*sizeof(int));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&didrhotx, Nt*Nx*sizeof(int));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&didthetatauq, Nq*Ntau*sizeof(int));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&didrhotauq, Nq*Ntau*sizeof(int));if (err!=0) callErr(hipGetErrorString(err));
	
	err=hipMalloc((void **)&demul, Nt*Nx*sizeof(float));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&dcosmul, Ntau*Nq*sizeof(float));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&dJ, Nt*Nx*sizeof(float));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&dreorids, Nt*Nx*sizeof(int));if (err!=0) callErr(hipGetErrorString(err));
	err=hipMalloc((void **)&dreoridsadj, Ntau*Nq*sizeof(int));if (err!=0) callErr(hipGetErrorString(err));

	err=hipMemcpy(ddthetatx,dthetatx, Nt*Nx*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	err=hipMemcpy(ddrhotx,drhotx, Nt*Nx*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	err=hipMemcpy(ddthetatauq,dthetatauq, Ntau*Nq*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	err=hipMemcpy(ddrhotauq,drhotauq, Ntau*Nq*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));

	err=hipMemcpy(dst,st,(Ntheta+2*add)*(Nrho+2*add)*sizeof(int),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	err=hipMemcpy(dstadj,stadj,(Ntheta+2*add)*(Nrho+2*add)*sizeof(int),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));


	err=hipMemcpy(dfZ,fZ,Ntheta_R2C*Nrho*sizeof(float2),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	err=hipMemcpy(didthetatx,idthetatx,Nt*Nx*sizeof(int),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	err=hipMemcpy(didrhotx,idrhotx,Nt*Nx*sizeof(int),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	err=hipMemcpy(didthetatauq,idthetatauq,Nq*Ntau*sizeof(int),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	err=hipMemcpy(didrhotauq,idrhotauq,Nq*Ntau*sizeof(int),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));

	err=hipMemcpy(demul,emul,Nt*Nx*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	err=hipMemcpy(dcosmul,cosmul,Nq*Ntau*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	err=hipMemcpy(dJ,J,Nt*Nx*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	err=hipMemcpy(dreorids,reorids,Nt*Nx*sizeof(int),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	err=hipMemcpy(dreoridsadj,reoridsadj,Ntau*Nq*sizeof(int),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));

	//fft plans for ni slices
	hipfftResult res1,res2;
	int ffts[]={Nrho,Ntheta};
	int idist = (Nrho+2*add)*(Ntheta+2*add);int odist = (Nrho)*((Ntheta/2+1));
	int inembed[] = {Nrho+2*add, Ntheta+2*add};int onembed[] = {Nrho, Ntheta/2+1};
	res1=hipfftPlanMany(&plan_forward, 2, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_R2C, ni);if (res1!=0) {char errs[16];sprintf(errs,"fwd hipfftPlanMany error %d",res1);callErr(errs);}
	res2=hipfftPlanMany(&plan_inverse, 2, ffts, onembed, 1, odist, inembed, 1, idist, HIPFFT_C2R, ni);if (res2!=0) {char errs[16];sprintf(errs,"inv hipfftPlanMany error %d",res1);callErr(errs);}
}
lpRgpu::~lpRgpu()
{
	hipFree(dftx);
	hipFree(dftauq);
	hipFree(dfl);
	hipFree(dflc);
	hipFree(dfZ);

	hipFree(dst);
	hipFree(dstadj);

	hipFree(ddthetatx);
	hipFree(ddrhotx);
	hipFree(ddthetatauq);
	hipFree(ddrhotauq);

	hipFree(didthetatx);
	hipFree(didrhotx);
	hipFree(didthetatauq);
	hipFree(didrhotauq);

	hipFree(demul);
	hipFree(dcosmul);
	hipFree(dJ);
	hipFree(dreorids);
	hipFree(dreoridsadj);
	
	hipfftDestroy(plan_forward);
	hipfftDestroy(plan_inverse);
}
void lpRgpu::getSizes(size_t *N)
{
	N[0]=Nt;N[1]=Nx;N[2]=Nq;N[3]=Ntau;N[4]=Ntheta;N[5]=Nrho;N[6]=ni;
}
void lpRgpu::fwd(float* out, int os1, int os2, float* in, int is1, int is2)
{
	err=hipMemcpy(dftx,in,Nt*Nx*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));	
	hipMemset(dfl,0,(Ntheta+2*add)*(Nrho+2*add)*ni*sizeof(float));

	int MBS31,MBS32,MBS33; MBS31=16;MBS32=16;MBS33=4;
	dim3 dimBlock(MBS31,MBS32,MBS33);	
	int MBS31g,MBS32g,MBS33g; MBS31g=8;MBS32g=8;MBS33g=8;
	dim3 dimBlockg(MBS31g,MBS32g,MBS33g);	
	
	
//mul erho,J
	int GS31=(int)ceil(Nt/(float)MBS31);int GS32=(int)ceil(Nx/(float)MBS32);int GS33=(int)ceil(ni/(float)MBS33);dim3 dimGrid0(GS31,GS32,GS33);	
	pmul <<<dimGrid0,dimBlock>>>(dftx,dJ,Nt,Nx,ni);hipDeviceSynchronize();
	pmul <<<dimGrid0,dimBlock>>>(dftx,demul,Nt,Nx,ni);hipDeviceSynchronize();	


//convtx
	GS31=(int)ceil(Ntheta/(float)MBS31g);GS32=(int)ceil(Nrho/(float)MBS32g);GS33=(int)ceil(ni/(float)MBS33g);dim3 dimGrid1(GS31,GS32,GS33);	
	convtx_ker<<<dimGrid1,dimBlockg>>>(dfl,dftx,dst,ddthetatx,ddrhotx,dreorids,Nt,Nx,Ntheta,Nrho,ni);hipDeviceSynchronize();

	
//fftlp
	hipfftExecR2C(plan_forward, (hipfftReal*)&dfl[(Ntheta+2*add)*add+add],(hipfftComplex*)dflc);hipDeviceSynchronize();
	GS31=(int)ceil(Ntheta_R2C/(float)MBS31);GS32=(int)ceil(Nrho/(float)MBS32);GS33=(int)ceil(ni/(float)MBS33);dim3 dimGrid2(GS31,GS32,GS33);
	mul<<<dimGrid2, dimBlock>>>(1/(float)(Ntheta*Nrho),dflc,dfZ,Ntheta_R2C,Nrho,ni);hipDeviceSynchronize();
	hipfftExecC2R(plan_inverse,(hipfftComplex*)dflc,(hipfftReal*)&dfl[(Ntheta+2*add)*add+add]);hipDeviceSynchronize();	

//convtauq
	GS31=(int)ceil(Nq/(float)MBS31);GS32=(int)ceil(Ntau/(float)MBS32);GS33=(int)ceil(ni/(float)MBS33);dim3 dimGrid3(GS31,GS32,GS33);	
	convtauq_ker<<<dimGrid3,dimBlock>>>(dftauq,dfl,didthetatauq,didrhotauq,ddthetatauq,ddrhotauq,dreoridsadj,Nq,Ntau,Ntheta,Nrho,ni);hipDeviceSynchronize();		

//mul cos
	pmul <<<dimGrid3,dimBlock>>>(dftauq,dcosmul,Nq,Ntau,ni);hipDeviceSynchronize();


	err=hipMemcpy(out,dftauq,Nq*Ntau*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));	
}
void lpRgpu::adj(float* out, int os1, int os2, float* in, int is1, int is2)
{
	err=hipMemcpy(dftauq,in,Ntau*Nq*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));	


	int MBS31,MBS32,MBS33; MBS31=16;MBS32=16;MBS33=4;
	dim3 dimBlock(MBS31,MBS32,MBS33);	
	int MBS31g,MBS32g,MBS33g; MBS31g=8;MBS32g=8;MBS33g=8;
	dim3 dimBlockg(MBS31g,MBS32g,MBS33g);	

	//mul cos
	int GS31=(int)ceil(Nq/(float)MBS31);int GS32=(int)ceil(Ntau/(float)MBS32);int GS33=(int)ceil(ni/(float)MBS33);dim3 dimGrid0(GS31,GS32,GS33);	
	pmul <<<dimGrid0,dimBlock>>>(dftauq,dcosmul,Nq,Ntau,ni);hipDeviceSynchronize();
	//conv
	hipMemset(dfl,0,(Ntheta+2*add)*(Nrho+2*add)*ni*sizeof(float));	
	GS31=(int)ceil(Ntheta/(float)MBS31g);GS32=(int)ceil(Nrho/(float)MBS32g);GS33=(int)ceil(ni/(float)MBS33g);dim3 dimGrid1(GS31,GS32,GS33);	
	convtx_ker<<<dimGrid1,dimBlockg>>>(dfl,dftauq,dstadj,ddthetatauq,ddrhotauq,dreoridsadj,Nq,Ntau,Ntheta,Nrho,ni);hipDeviceSynchronize();
	//fftlp
	hipfftExecR2C(plan_forward, (hipfftReal*)dfl,(hipfftComplex*)dflc);	
	GS31=(int)ceil(Ntheta_R2C/(float)MBS31);GS32=(int)ceil(Nrho/(float)MBS32);GS33=(int)ceil(ni/(float)MBS33);dim3 dimGrid2(GS31,GS32,GS33);
	muladj<<<dimGrid2, dimBlock>>>(1/(float)(Ntheta*Nrho),dflc,dfZ,Ntheta_R2C,Nrho,ni);hipDeviceSynchronize();
	hipfftExecC2R(plan_inverse,(hipfftComplex*)dflc,(hipfftReal*)dfl);hipDeviceSynchronize();
	//conv	
	GS31=(int)ceil(Nt/(float)MBS31);GS32=(int)ceil(Nx/(float)MBS32);GS33=(int)ceil(ni/(float)MBS33);dim3 dimGrid3(GS31,GS32,GS33);	
	convtauq_ker<<<dimGrid3,dimBlock>>>(dftx,dfl,didthetatx,didrhotx,ddthetatx,ddrhotx,dreorids,Nt,Nx,Ntheta,Nrho,ni);hipDeviceSynchronize();
	//mul erho,J
	pmul <<<dimGrid3,dimBlock>>>(dftx,dJ,Nt,Nx,ni);hipDeviceSynchronize();
	pmul <<<dimGrid3,dimBlock>>>(dftx,demul,Nt,Nx,ni);hipDeviceSynchronize();	

	err=hipMemcpy(out,dftx,Nt*Nx*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));	
}

void lpRgpu::fftlp(float* out,float* in)
{
	err=hipMemcpy(dfl,in,Ntheta*Nrho*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
	//forward FFT
	hipfftExecR2C(plan_forward, (hipfftReal*)dfl,(hipfftComplex*)dflc);
	//multiplication by fZ
		int MBS31,MBS32,MBS33; MBS31=8;MBS32=8;MBS33=16;
	dim3 dimBlock(MBS31,MBS32,MBS33);	
	int GS31=(int)ceil(Ntheta_R2C/(float)MBS31);int GS32=(int)ceil(Nrho/(float)MBS32);int GS33=(int)ceil(ni/(float)MBS33);dim3 dimGrid(GS31,GS32,GS33);
	mul<<<dimGrid, dimBlock>>>(1/(float)(Ntheta*Nrho),dflc,dfZ,Ntheta_R2C,Nrho,ni);hipDeviceSynchronize();
	//inverse FFT
	hipfftExecC2R(plan_inverse,(hipfftComplex*)dflc,(hipfftReal*)dfl);
	hipDeviceSynchronize();

	err=hipMemcpy(out,dfl,Ntheta*Nrho*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
}
void lpRgpu::fftlpadj(float* out,float* in)
{
	//forward FFT	
	hipfftExecR2C(plan_forward, (hipfftReal*)dfl,(hipfftComplex*)dflc);
	//multiplication by fZ
	int MBS31,MBS32,MBS33; MBS31=8;MBS32=8;MBS33=16;
	dim3 dimBlock(MBS31,MBS32,MBS33);	
	int GS31=(int)ceil(Ntheta_R2C/(float)MBS31);int GS32=(int)ceil(Nrho/(float)MBS32);int GS33=(int)ceil(ni/(float)MBS33);dim3 dimGrid(GS31,GS32,GS33);
	muladj<<<dimGrid, dimBlock>>>(1/(float)(Ntheta*Nrho),dflc,dfZ,Ntheta_R2C,Nrho,ni);hipDeviceSynchronize();
	//inverse FFT
	hipfftExecC2R(plan_inverse,(hipfftComplex*)dflc,(hipfftReal*)dfl);
	hipDeviceSynchronize();	
	err=hipMemcpy(out,dfl,Ntheta*Nrho*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));
}
void lpRgpu::convtx(float* out,float* in)
{
	err=hipMemcpy(dftx,in,Nt*Nx*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));	
	hipMemset(dfl,0,(Ntheta+2*add)*(Nrho+2*add)*ni*sizeof(float));
	int MBS31_,MBS32_,MBS33_;
	MBS31_=32;MBS32_=32;MBS33_=1;
	dim3 dimBlock(MBS31_,MBS32_,MBS33_);	
	int GS31=(int)ceil(Ntheta/(float)MBS31_);int GS32=(int)ceil(Nrho/(float)MBS32_);int GS33=(int)ceil(ni/(float)MBS33_);dim3 dimGrid(GS31,GS32,GS33);	
	convtx_ker<<<dimGrid,dimBlock>>>(dfl,dftx,dst,ddthetatx,ddrhotx,dreorids,Nt,Nx,Ntheta,Nrho,ni);hipDeviceSynchronize();
	err=hipMemcpy(out,dfl,Ntheta*Nrho*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));	
}
void lpRgpu::convtauq(float* out,float* in)
{
	err=hipMemcpy(dfl,in,(Ntheta+2*add)*(Nrho+2*add)*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));	
	int MBS31,MBS32,MBS33; MBS31=8;MBS32=8;MBS33=16;
	dim3 dimBlock(MBS31,MBS32,MBS33);	
	int GS31=(int)ceil(Nq/(float)MBS31);int GS32=(int)ceil(Ntau/(float)MBS32);int GS33=(int)ceil(ni/(float)MBS33);dim3 dimGrid(GS31,GS32,GS33);	
	convtauq_ker<<<dimGrid,dimBlock>>>(dftauq,dfl,didthetatauq,didrhotauq,ddthetatauq,ddrhotauq,dreoridsadj,Nq,Ntau,Ntheta,Nrho,ni);hipDeviceSynchronize();
	err=hipMemcpy(out,dftauq,Nq*Ntau*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));	
}

void lpRgpu::convtauqadj(float* out,float* in)
{
	err=hipMemcpy(dftauq,in,Ntau*Nq*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));	
	hipMemset(dfl,0,(Ntheta+2*add)*(Nrho+2*add)*ni*sizeof(float));
	int MBS31_,MBS32_,MBS33_;
	MBS31_=8;MBS32_=8;MBS33_=16;
	dim3 dimBlock(MBS31_,MBS32_,MBS33_);	
	int GS31=(int)ceil(Ntheta/(float)MBS31_);int GS32=(int)ceil(Nrho/(float)MBS32_);int GS33=(int)ceil(ni/(float)MBS33_);dim3 dimGrid(GS31,GS32,GS33);	
	convtx_ker<<<dimGrid,dimBlock>>>(dfl,dftauq,dstadj,ddthetatauq,ddrhotauq,dreorids,Nq,Ntau,Ntheta,Nrho,ni);hipDeviceSynchronize();
	err=hipMemcpy(out,dfl,Ntheta*Nrho*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));	
}

void lpRgpu::convtxadj(float* out,float* in)
{
	err=hipMemcpy(dfl,in,(Ntheta+2*add)*(Nrho+2*add)*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));	
		int MBS31,MBS32,MBS33; MBS31=8;MBS32=8;MBS33=16;
	dim3 dimBlock(MBS31,MBS32,MBS33);	
	int GS31=(int)ceil(Nt/(float)MBS31);int GS32=(int)ceil(Nx/(float)MBS32);int GS33=(int)ceil(ni/(float)MBS33);dim3 dimGrid(GS31,GS32,GS33);	
	convtauq_ker<<<dimGrid,dimBlock>>>(dftx,dfl,didthetatx,didrhotx,ddthetatx,ddrhotx,dreoridsadj,Nt,Nx,Ntheta,Nrho,ni);hipDeviceSynchronize();
	err=hipMemcpy(out,dftx,Nt*Nx*ni*sizeof(float),hipMemcpyDefault);if (err!=0) callErr(hipGetErrorString(err));	
}

void lpRgpu::printCurrentGPUMemory(const char* str)
{
	size_t gpufree1,gputotal;
	hipMemGetInfo(&gpufree1,&gputotal);
	if(str!=NULL)
		printf("%s gpufree=%.0fM,gputotal=%.0fM\n",str,gpufree1/(float)(1024*1024),gputotal/(float)(1024*1024));
	else
		printf("gpufree=%.0fM,gputotal=%.0fM\n",gpufree1/(float)(1024*1024),gputotal/(float)(1024*1024));
}
